#include "asteroid/renderer/renderer.h"
#include "asteroid/renderer/path_tracer.h"
#include "asteroid/util/macro.h"

using namespace Asteroid;

void Renderer::OnResize(unsigned int width, unsigned int height) {
    if (m_FinalImage) {
        // No resize necessary
        if (m_FinalImage->GetWidth() == width && m_FinalImage->GetHeight() == height)
            return;

        m_FinalImage->Resize(width, height);
    } else {
        m_FinalImage = std::make_shared<Image>(width, height);
    }

    auto pixel_num = width * height;

    hipFree(m_Rays);
    hipMalloc((void **) &m_Rays, sizeof(Ray) * pixel_num);

    hipFree(m_Intersections);
    hipMalloc((void **) &m_Intersections, sizeof(Intersection) * pixel_num);

    hipFree(m_AccumulationData);
    hipMalloc((void **) &m_AccumulationData, sizeof(glm::vec4) * pixel_num);
    hipMemset(m_AccumulationData, 0, sizeof(glm::vec4) * pixel_num);

    hipFree(m_ImageData);
    hipMalloc((void **) &m_ImageData, sizeof(glm::u8vec4) * pixel_num);
}

void Renderer::Render(const Scene &scene, const Camera &camera) {
    auto width = m_FinalImage->GetWidth();
    auto height = m_FinalImage->GetHeight();

    auto sceneView = SceneView(scene);

    // Execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(width / block.x, height / block.y, 1);

    GeneratePrimaryRay<<<grid, block>>>(camera, m_Rays);
    CUDA_SYNC_CHECK()

    hipMemset(m_AccumulationData, 0, sizeof(glm::vec4) * width * height);

    int bounces = 2;
    for (int i = 0; i < bounces; i++) {
        ComputeIntersection<<<grid, block>>>(sceneView, m_Rays, width, height, m_Intersections);
        CUDA_SYNC_CHECK()

        Shading<<<grid, block>>>(m_Rays, m_Intersections, m_AccumulationData, width, height);
        CUDA_SYNC_CHECK()
    }

    ConvertToRGBA<<<grid, block>>>(m_AccumulationData, width, height, m_ImageData);

    m_FinalImage->SetData(m_ImageData);
}
