#include "asteroid/renderer/renderer.h"
#include "asteroid/renderer/path_tracer.h"
#include "asteroid/util/macro.h"

using namespace Asteroid;

void Renderer::OnResize(unsigned int width, unsigned int height)
{
	if (m_FinalImage)
	{
		// No resize necessary
		if (m_FinalImage->GetWidth() == width && m_FinalImage->GetHeight() == height)
			return;

		m_FinalImage->Resize(width, height);
	}
	else
	{
		m_FinalImage = std::make_shared<Image>(width, height);
	}

	auto pixel_num = width * height;

	hipFree(m_ImageData);
    hipMalloc((void**)&m_ImageData, sizeof(glm::u8vec4) * pixel_num);

    hipFree(m_AccumulationData);
    hipMalloc((void**)&m_AccumulationData, sizeof(glm::vec4) * pixel_num);

    hipFree(m_Rays);
    hipMalloc((void**)&m_Rays, sizeof(Ray) * pixel_num);

    hipFree(m_Intersections);
    hipMalloc((void**)&m_Intersections, sizeof(Intersection) * pixel_num);
}

void Renderer::Render(const Scene& scene, const Camera& camera)
{
	auto width = m_FinalImage->GetWidth();
	auto height = m_FinalImage->GetHeight();

	auto sceneView = SceneView(scene);

	// Execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(width / block.x, height / block.y, 1);

    GeneratePrimaryRay<<<grid, block>>>(camera, m_Rays);
    CUDA_SYNC_CHECK()

    ComputeIntersection<<<grid, block>>>(sceneView, m_Rays, width, height,m_Intersections);
    CUDA_SYNC_CHECK()

    PerPixel<<<grid, block>>>(sceneView, m_Rays, m_ImageData, width, height);
    CUDA_SYNC_CHECK()

	m_FinalImage->SetData(m_ImageData);
}
