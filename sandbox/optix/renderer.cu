#include "hip/hip_runtime.h"
#include "renderer.h"

// this include may only appear in a single source file:
#include <optix_function_table_definition.h>

#include "asteroid/util/log.h"
#include "otkHelloKernelPTX.h"

using namespace Asteroid;

//extern "C" char embedded_ptx_code[];

static void context_log_cb(unsigned int level,
                           const char *tag,
                           const char *message,
                           void *)
{
    fprintf(stderr, "[%2d][%12s]: %s\n", (int) level, tag, message);
}

/*! SBT record for a raygen program */
struct __align__(OPTIX_SBT_RECORD_ALIGNMENT) RaygenRecord
{
    __align__(OPTIX_SBT_RECORD_ALIGNMENT) char header[OPTIX_SBT_RECORD_HEADER_SIZE];
    // just a dummy value - later examples will use more interesting
    // data here
    void *data;
};

/*! SBT record for a miss program */
struct __align__(OPTIX_SBT_RECORD_ALIGNMENT) MissRecord
{
    __align__(OPTIX_SBT_RECORD_ALIGNMENT) char header[OPTIX_SBT_RECORD_HEADER_SIZE];
    // just a dummy value - later examples will use more interesting
    // data here
    void *data;
};

/*! SBT record for a hitgroup program */
struct __align__(OPTIX_SBT_RECORD_ALIGNMENT) HitgroupRecord
{
    __align__(OPTIX_SBT_RECORD_ALIGNMENT) char header[OPTIX_SBT_RECORD_HEADER_SIZE];
    // just a dummy value - later examples will use more interesting
    // data here
    int objectID;
};

Renderer::Renderer()
{
    initOptix();

    AST_CORE_INFO("creating optix context ...");
    createContext();

    AST_CORE_INFO("creating optix module ...");
    createModule();

    AST_CORE_INFO("create optix raygen program ...");
    createRaygenPG();

    AST_CORE_INFO("creating optix miss program ...");
    createMissPG();

    AST_CORE_INFO("creating optix hit group program ...");
    createHitGroupPG();

    AST_CORE_INFO("creating optix pipeline ...");
    createPipeline();

    AST_CORE_INFO("creating optix shader binding table ...");
    createSBT();

    m_launchParamsBuffer.alloc(sizeof(m_launchParams));
}

void Renderer::OnResize(unsigned int width, unsigned int height)
{
    if (m_finalImage)
    {
        // No resize necessary
        if (m_finalImage->GetWidth() == width && m_finalImage->GetHeight() == height)
            return;

        m_finalImage->Resize(width, height);
    } else
    {
        m_finalImage = std::make_shared<Image>(width, height);
    }

    m_colorBuffer.resize(width * height * sizeof(unsigned int));

    m_launchParams.width = width;
    m_launchParams.height = height;
    m_launchParams.colorBuffer = (unsigned int *) m_colorBuffer.devicePtr();
}

void Renderer::Render()
{
    if (m_launchParams.width == 0) return;

    m_launchParamsBuffer.upload(&m_launchParams,1);
    m_launchParams.frameID++;

    AST_OPTIX_CHECK(optixLaunch(/*! pipeline we're launching launch: */
        m_pipeline,m_stream,
        /*! parameters and SBT */
        (hipDeviceptr_t)m_launchParamsBuffer.devicePtr(),
        m_launchParamsBuffer.m_sizeInBytes,
        &m_sbt,
        /*! dimensions of the launch: */
        m_launchParams.width,
        m_launchParams.height,
        1
    ));
    // sync - make sure the frame is rendered before we download and
    // display (obviously, for a high-performance application you
    // want to use streams and double-buffering, but for this simple
    // example, this will have to do)
    AST_CUDA_SYNC_CHECK();

    m_finalImage->SetData(m_colorBuffer.devicePtr());
}

void Renderer::initOptix()
{
    AST_CORE_INFO("init optix ...");

    hipFree(nullptr);
    int numDevices;
    hipGetDeviceCount(&numDevices);
    if (numDevices == 0)
    {
        AST_CORE_ERROR("no CUDA capable devices found!");
    }

    AST_CORE_INFO("found {0} CUDA capable devices", numDevices);

    AST_OPTIX_CHECK(optixInit());

    AST_CORE_INFO("initialize optix successfully!");
}

void Renderer::createContext()
{
    const int deviceID = 0;
    AST_CUDA_CHECK(hipSetDevice(deviceID));
    AST_CUDA_CHECK(hipStreamCreate(&m_stream));

    hipGetDeviceProperties(&m_deviceProps, deviceID);
    AST_CORE_INFO("running on device: {0}", m_deviceProps.name);

    m_cudaContext = 0;
//    auto cuRes = hipCtxGetCurrent(&m_cudaContext);
//    if (cuRes != hipSuccess)
//        fprintf(stderr, "Error querying current context: error code %d\n", cuRes);

    AST_OPTIX_CHECK(optixDeviceContextCreate(m_cudaContext, 0, &m_optixContext));
    AST_OPTIX_CHECK(optixDeviceContextSetLogCallback
                        (m_optixContext, context_log_cb, nullptr, 4));
}

void Renderer::createModule()
{
    m_moduleCompileOptions.maxRegisterCount = OPTIX_COMPILE_DEFAULT_MAX_REGISTER_COUNT;
    m_moduleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT;
    m_moduleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_NONE;

    m_pipelineCompileOptions.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
    m_pipelineCompileOptions.usesMotionBlur = false;
    m_pipelineCompileOptions.numPayloadValues = 2;
    m_pipelineCompileOptions.numAttributeValues = 2;
    m_pipelineCompileOptions.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
    m_pipelineCompileOptions.pipelineLaunchParamsVariableName = "optixLaunchParams";

//    const std::string ptxCode = embedded_ptx_code;

    char log[2048];
    auto sizeof_log = sizeof(log);
    AST_OPTIX_CHECK(optixModuleCreate(m_optixContext,
                                             &m_moduleCompileOptions,
                                             &m_pipelineCompileOptions,
                                             program_ptx_text(),
                                             program_ptx_size,
                                             log,
                                             &sizeof_log,
                                             &m_module));
    if (sizeof_log > 1)
        fprintf(stderr, "Log:\n%s\n", log);
}

void Renderer::createRaygenPG()
{
    m_raygenPGs.resize(1);

    OptixProgramGroupOptions pgOptions = {};
    OptixProgramGroupDesc pgDesc = {};
    pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
    pgDesc.raygen.module = m_module;
    pgDesc.raygen.entryFunctionName = "__raygen__renderFrame";

    char log[2048];
    auto sizeof_log = sizeof(log);
    AST_OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
                                            &pgDesc,
                                            1,
                                            &pgOptions,
                                            log,
                                            &sizeof_log,
                                            &m_raygenPGs[0]));
    if (sizeof_log > 1)
        fprintf(stderr, "Log:\n%s\n", log);
}

void Renderer::createMissPG()
{
    m_missPGs.resize(1);
    OptixProgramGroupOptions pgOptions = {};
    OptixProgramGroupDesc pgDesc = {};
    pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
    pgDesc.miss.module = m_module;
    pgDesc.miss.entryFunctionName = "__miss__radiance";

    char log[2048];
    auto sizeof_log = sizeof(log);
    AST_OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
                                            &pgDesc,
                                            1,
                                            &pgOptions,
                                            log,
                                            &sizeof_log,
                                            &m_missPGs[0]));
    if (sizeof_log > 1)
        fprintf(stderr, "Log:\n%s\n", log);
}

void Renderer::createHitGroupPG()
{
    m_hitgroupPGs.resize(1);
    OptixProgramGroupOptions pgOptions = {};
    OptixProgramGroupDesc pgDesc = {};
    pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
    pgDesc.hitgroup.moduleCH = m_module;
    pgDesc.hitgroup.entryFunctionNameCH = "__closesthit__radiance";
    pgDesc.hitgroup.moduleAH = m_module;
    pgDesc.hitgroup.entryFunctionNameAH = "__anyhit__radiance";

    char log[2048];
    auto sizeof_log = sizeof(log);
    AST_OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
                                            &pgDesc,
                                            1,
                                            &pgOptions,
                                            log,
                                            &sizeof_log,
                                            &m_hitgroupPGs[0]));
    if (sizeof_log > 1)
        fprintf(stderr, "Log:\n%s\n", log);
}

void Renderer::createPipeline()
{
    std::vector<OptixProgramGroup> programGroups;
    programGroups.insert(programGroups.end(), m_raygenPGs.begin(), m_raygenPGs.end());
    programGroups.insert(programGroups.end(), m_missPGs.begin(), m_missPGs.end());
    programGroups.insert(programGroups.end(), m_hitgroupPGs.begin(), m_hitgroupPGs.end());

    char log[2048];
    auto sizeof_log = sizeof(log);
    AST_OPTIX_CHECK(optixPipelineCreate(m_optixContext,
                                        &m_pipelineCompileOptions,
                                        &m_pipelineLinkOptions,
                                        programGroups.data(),
                                        programGroups.size(),
                                        log,
                                        &sizeof_log,
                                        &m_pipeline));
    if (sizeof_log > 1)
        fprintf(stderr, "Log:\n%s\n", log);
}

void Renderer::createSBT()
{
    std::vector<RaygenRecord> raygenRecords;
    for (int i = 0; i < m_raygenPGs.size(); i++)
    {
        RaygenRecord rec{};
        AST_OPTIX_CHECK(optixSbtRecordPackHeader(m_raygenPGs[i], &rec));
        rec.data = nullptr; /* for now ... */
        raygenRecords.push_back(rec);
    }
    m_raygenRecords.allocAndUpload(raygenRecords);
    m_sbt.raygenRecord = (hipDeviceptr_t)m_raygenRecords.m_devicePtr;

    std::vector<MissRecord> missRecords;
    for (int i = 0; i < m_missPGs.size(); i++)
    {
        MissRecord rec{};
        AST_OPTIX_CHECK(optixSbtRecordPackHeader(m_missPGs[i], &rec));
        rec.data = nullptr; /* for now ... */
        missRecords.push_back(rec);
    }
    m_missRecords.allocAndUpload(missRecords);
    m_sbt.missRecordBase = (hipDeviceptr_t)m_missRecords.devicePtr();
    m_sbt.missRecordStrideInBytes = sizeof(MissRecord);
    m_sbt.missRecordCount = static_cast<int>(missRecords.size());

    std::vector<HitgroupRecord> hitgroupRecords;
    for (int i = 0; i < m_hitgroupPGs.size(); i++)
    {
        HitgroupRecord rec{};
        AST_OPTIX_CHECK(optixSbtRecordPackHeader(m_hitgroupPGs[i], &rec));
        rec.objectID = i; /* for now ... */
        hitgroupRecords.push_back(rec);
    }
    m_hitgroupRecords.allocAndUpload(hitgroupRecords);
    m_sbt.hitgroupRecordBase = (hipDeviceptr_t)m_hitgroupRecords.devicePtr();
    m_sbt.hitgroupRecordStrideInBytes = sizeof(HitgroupRecord);
    m_sbt.hitgroupRecordCount = static_cast<int>(hitgroupRecords.size());
}