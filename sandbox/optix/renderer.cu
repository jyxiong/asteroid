#include "hip/hip_runtime.h"
#include "renderer.h"

// this include may only appear in a single source file:
#include <optix_function_table_definition.h>

#include "asteroid/util/log.h"
#include "otkHelloKernelPTX.h"

using namespace Asteroid;

//extern "C" char embedded_ptx_code[];

void TriangleMesh::addCube(const float3 &center, const float3 &size) {
    auto transform = Matrix4x4::translate(center - 0.5f * size) *
                     Matrix4x4::scale(size);
    addUnitCube(transform);
}

/*! add a unit cube (subject to given xfm matrix) to the current
    triangleMesh */
void TriangleMesh::addUnitCube(const Matrix4x4 &transform) {
    int firstVertexID = (int) vertex.size();
    vertex.push_back(make_float3(transform * make_float4(0.f, 0.f, 0.f, 1.f)));
    vertex.push_back(make_float3(transform * make_float4(1.f, 0.f, 0.f, 1.f)));
    vertex.push_back(make_float3(transform * make_float4(0.f, 1.f, 0.f, 1.f)));
    vertex.push_back(make_float3(transform * make_float4(1.f, 1.f, 0.f, 1.f)));
    vertex.push_back(make_float3(transform * make_float4(0.f, 0.f, 1.f, 1.f)));
    vertex.push_back(make_float3(transform * make_float4(1.f, 0.f, 1.f, 1.f)));
    vertex.push_back(make_float3(transform * make_float4(0.f, 1.f, 1.f, 1.f)));
    vertex.push_back(make_float3(transform * make_float4(1.f, 1.f, 1.f, 1.f)));


    int indices[] = {0, 1, 3, 2, 3, 0,
                     5, 7, 6, 5, 6, 4,
                     0, 4, 5, 0, 5, 1,
                     2, 3, 7, 2, 7, 6,
                     1, 5, 7, 1, 7, 3,
                     4, 0, 2, 4, 2, 6
    };
    for (int i = 0; i < 12; i++)
        index.push_back(firstVertexID + make_int3(indices[3 * i + 0],
                                                  indices[3 * i + 1],
                                                  indices[3 * i + 2]));
}


static void context_log_cb(unsigned int level,
                           const char *tag,
                           const char *message,
                           void *) {
    fprintf(stderr, "[%2d][%12s]: %s\n", (int) level, tag, message);
}

/*! SBT record for a raygen program */
struct __align__(OPTIX_SBT_RECORD_ALIGNMENT) RaygenRecord {
    __align__(OPTIX_SBT_RECORD_ALIGNMENT) char header[OPTIX_SBT_RECORD_HEADER_SIZE];
    // just a dummy value - later examples will use more interesting
    // data here
    void *data;
};

/*! SBT record for a miss program */
struct __align__(OPTIX_SBT_RECORD_ALIGNMENT) MissRecord {
    __align__(OPTIX_SBT_RECORD_ALIGNMENT) char header[OPTIX_SBT_RECORD_HEADER_SIZE];
    // just a dummy value - later examples will use more interesting
    // data here
    void *data;
};

/*! SBT record for a hitgroup program */
struct __align__(OPTIX_SBT_RECORD_ALIGNMENT) HitgroupRecord {
    __align__(OPTIX_SBT_RECORD_ALIGNMENT) char header[OPTIX_SBT_RECORD_HEADER_SIZE];
    // just a dummy value - later examples will use more interesting
    // data here
    int objectID;
};

Renderer::Renderer() {
    initOptix();

    AST_CORE_INFO("creating optix context ...");
    createContext();

    AST_CORE_INFO("creating optix module ...");
    createModule();

    AST_CORE_INFO("create optix raygen program ...");
    createRaygenPG();

    AST_CORE_INFO("creating optix miss program ...");
    createMissPG();

    AST_CORE_INFO("creating optix hit group program ...");
    createHitGroupPG();

    AST_CORE_INFO("creating optix pipeline ...");
    createPipeline();

    AST_CORE_INFO("creating optix shader binding table ...");
    createSBT();

    m_launchParamsBuffer.alloc(sizeof(m_launchParams));
}

void Renderer::OnResize(unsigned int width, unsigned int height) {
    if (m_finalImage) {
        // No resize necessary
        if (m_finalImage->GetWidth() == width && m_finalImage->GetHeight() == height)
            return;

        m_finalImage->Resize(width, height);
    } else {
        m_finalImage = std::make_shared<Image>(width, height);
    }

    m_colorBuffer.resize(width * height * sizeof(unsigned int));

    m_launchParams.frame.size = make_int2(width, height);
    m_launchParams.frame.colorBuffer = (unsigned int *) m_colorBuffer.devicePtr();
}

void Renderer::Render() {
    if (m_launchParams.frame.size.x == 0) return;

    m_launchParamsBuffer.upload(&m_launchParams, 1);

    AST_OPTIX_CHECK(optixLaunch(/*! pipeline we're launching launch: */
        m_pipeline, m_stream,
        /*! parameters and SBT */
        (hipDeviceptr_t) m_launchParamsBuffer.devicePtr(),
        m_launchParamsBuffer.m_sizeInBytes,
        &m_sbt,
        /*! dimensions of the launch: */
        m_launchParams.frame.size.x,
        m_launchParams.frame.size.y,
        1
    ));
    // sync - make sure the frame is rendered before we download and
    // display (obviously, for a high-performance application you
    // want to use streams and double-buffering, but for this simple
    // example, this will have to do)
    AST_CUDA_SYNC_CHECK();

    m_finalImage->SetData(m_colorBuffer.devicePtr());
}

void Renderer::setCamera(const Camera &camera)
{
    m_launchParams.camera.position  = camera.position;
    m_launchParams.camera.direction = camera.direction;
//    const float cosFovy = 0.66f;
//    const float aspect = m_launchParams.frame.size.x / float(m_launchParams.frame.size.y);
//    m_launchParams.camera.horizontal
//        = cosFovy * aspect * normalize(cross(m_launchParams.camera.direction,
//                                             camera.up));
//    m_launchParams.camera.vertical
//        = cosFovy * normalize(cross(m_launchParams.camera.horizontal,
//                                    m_launchParams.camera.direction));
}

void Renderer::initOptix() {
    AST_CORE_INFO("init optix ...");

    hipFree(nullptr);
    int numDevices;
    hipGetDeviceCount(&numDevices);
    if (numDevices == 0) {
        AST_CORE_ERROR("no CUDA capable devices found!");
    }

    AST_CORE_INFO("found {0} CUDA capable devices", numDevices);

    AST_OPTIX_CHECK(optixInit());

    AST_CORE_INFO("initialize optix successfully!");
}

void Renderer::createContext() {
    const int deviceID = 0;
    AST_CUDA_CHECK(hipSetDevice(deviceID));
    AST_CUDA_CHECK(hipStreamCreate(&m_stream));

    hipGetDeviceProperties(&m_deviceProps, deviceID);
    AST_CORE_INFO("running on device: {0}", m_deviceProps.name);

    m_cudaContext = 0;
//    auto cuRes = hipCtxGetCurrent(&m_cudaContext);
//    if (cuRes != hipSuccess)
//        fprintf(stderr, "Error querying current context: error code %d\n", cuRes);

    AST_OPTIX_CHECK(optixDeviceContextCreate(m_cudaContext, 0, &m_optixContext));
    AST_OPTIX_CHECK(optixDeviceContextSetLogCallback
                        (m_optixContext, context_log_cb, nullptr, 4));
}

void Renderer::createModule() {
    m_moduleCompileOptions.maxRegisterCount = OPTIX_COMPILE_DEFAULT_MAX_REGISTER_COUNT;
    m_moduleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT;
    m_moduleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_NONE;

    m_pipelineCompileOptions.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
    m_pipelineCompileOptions.usesMotionBlur = false;
    m_pipelineCompileOptions.numPayloadValues = 2;
    m_pipelineCompileOptions.numAttributeValues = 2;
    m_pipelineCompileOptions.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
    m_pipelineCompileOptions.pipelineLaunchParamsVariableName = "optixLaunchParams";

//    const std::string ptxCode = embedded_ptx_code;

    char log[2048];
    auto sizeof_log = sizeof(log);
    AST_OPTIX_CHECK(optixModuleCreate(m_optixContext,
                                      &m_moduleCompileOptions,
                                      &m_pipelineCompileOptions,
                                      program_ptx_text(),
                                      program_ptx_size,
                                      log,
                                      &sizeof_log,
                                      &m_module));
    if (sizeof_log > 1)
        fprintf(stderr, "Log:\n%s\n", log);
}

void Renderer::createRaygenPG() {
    m_raygenPGs.resize(1);

    OptixProgramGroupOptions pgOptions = {};
    OptixProgramGroupDesc pgDesc = {};
    pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
    pgDesc.raygen.module = m_module;
    pgDesc.raygen.entryFunctionName = "__raygen__renderFrame";

    char log[2048];
    auto sizeof_log = sizeof(log);
    AST_OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
                                            &pgDesc,
                                            1,
                                            &pgOptions,
                                            log,
                                            &sizeof_log,
                                            &m_raygenPGs[0]));
    if (sizeof_log > 1)
        fprintf(stderr, "Log:\n%s\n", log);
}

void Renderer::createMissPG() {
    m_missPGs.resize(1);
    OptixProgramGroupOptions pgOptions = {};
    OptixProgramGroupDesc pgDesc = {};
    pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
    pgDesc.miss.module = m_module;
    pgDesc.miss.entryFunctionName = "__miss__radiance";

    char log[2048];
    auto sizeof_log = sizeof(log);
    AST_OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
                                            &pgDesc,
                                            1,
                                            &pgOptions,
                                            log,
                                            &sizeof_log,
                                            &m_missPGs[0]));
    if (sizeof_log > 1)
        fprintf(stderr, "Log:\n%s\n", log);
}

void Renderer::createHitGroupPG() {
    m_hitgroupPGs.resize(1);
    OptixProgramGroupOptions pgOptions = {};
    OptixProgramGroupDesc pgDesc = {};
    pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
    pgDesc.hitgroup.moduleCH = m_module;
    pgDesc.hitgroup.entryFunctionNameCH = "__closesthit__radiance";
    pgDesc.hitgroup.moduleAH = m_module;
    pgDesc.hitgroup.entryFunctionNameAH = "__anyhit__radiance";

    char log[2048];
    auto sizeof_log = sizeof(log);
    AST_OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
                                            &pgDesc,
                                            1,
                                            &pgOptions,
                                            log,
                                            &sizeof_log,
                                            &m_hitgroupPGs[0]));
    if (sizeof_log > 1)
        fprintf(stderr, "Log:\n%s\n", log);
}

void Renderer::createPipeline() {
    std::vector<OptixProgramGroup> programGroups;
    programGroups.insert(programGroups.end(), m_raygenPGs.begin(), m_raygenPGs.end());
    programGroups.insert(programGroups.end(), m_missPGs.begin(), m_missPGs.end());
    programGroups.insert(programGroups.end(), m_hitgroupPGs.begin(), m_hitgroupPGs.end());

    char log[2048];
    auto sizeof_log = sizeof(log);
    AST_OPTIX_CHECK(optixPipelineCreate(m_optixContext,
                                        &m_pipelineCompileOptions,
                                        &m_pipelineLinkOptions,
                                        programGroups.data(),
                                        programGroups.size(),
                                        log,
                                        &sizeof_log,
                                        &m_pipeline));
    if (sizeof_log > 1)
        fprintf(stderr, "Log:\n%s\n", log);
}

void Renderer::createSBT() {
    std::vector<RaygenRecord> raygenRecords;
    for (int i = 0; i < m_raygenPGs.size(); i++) {
        RaygenRecord rec{};
        AST_OPTIX_CHECK(optixSbtRecordPackHeader(m_raygenPGs[i], &rec));
        rec.data = nullptr; /* for now ... */
        raygenRecords.push_back(rec);
    }
    m_raygenRecords.allocAndUpload(raygenRecords);
    m_sbt.raygenRecord = (hipDeviceptr_t) m_raygenRecords.m_devicePtr;

    std::vector<MissRecord> missRecords;
    for (int i = 0; i < m_missPGs.size(); i++) {
        MissRecord rec{};
        AST_OPTIX_CHECK(optixSbtRecordPackHeader(m_missPGs[i], &rec));
        rec.data = nullptr; /* for now ... */
        missRecords.push_back(rec);
    }
    m_missRecords.allocAndUpload(missRecords);
    m_sbt.missRecordBase = (hipDeviceptr_t) m_missRecords.devicePtr();
    m_sbt.missRecordStrideInBytes = sizeof(MissRecord);
    m_sbt.missRecordCount = static_cast<int>(missRecords.size());

    std::vector<HitgroupRecord> hitgroupRecords;
    for (int i = 0; i < m_hitgroupPGs.size(); i++) {
        HitgroupRecord rec{};
        AST_OPTIX_CHECK(optixSbtRecordPackHeader(m_hitgroupPGs[i], &rec));
        rec.objectID = i; /* for now ... */
        hitgroupRecords.push_back(rec);
    }
    m_hitgroupRecords.allocAndUpload(hitgroupRecords);
    m_sbt.hitgroupRecordBase = (hipDeviceptr_t) m_hitgroupRecords.devicePtr();
    m_sbt.hitgroupRecordStrideInBytes = sizeof(HitgroupRecord);
    m_sbt.hitgroupRecordCount = static_cast<int>(hitgroupRecords.size());
}

OptixTraversableHandle Renderer::createAccel() {
    // upload the model to the device: the builder
    m_vertexBuffer.allocAndUpload(m_model.vertex);
    m_indexBuffer.allocAndUpload(m_model.index);

    OptixTraversableHandle asHandle{0};

    // ==================================================================
    // triangle inputs
    // ==================================================================
    OptixBuildInput triangleInput = {};
    triangleInput.type
        = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;

    // create local variables, because we need a *pointer* to the
    // device pointers
    auto d_vertices = (hipDeviceptr_t) m_vertexBuffer.devicePtr();
    auto d_indices = (hipDeviceptr_t) m_indexBuffer.devicePtr();

    triangleInput.triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
    triangleInput.triangleArray.vertexStrideInBytes = sizeof(float3);
    triangleInput.triangleArray.numVertices = (int) m_model.vertex.size();
    triangleInput.triangleArray.vertexBuffers = &d_vertices;

    triangleInput.triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
    triangleInput.triangleArray.indexStrideInBytes = sizeof(int3);
    triangleInput.triangleArray.numIndexTriplets = (int) m_model.index.size();
    triangleInput.triangleArray.indexBuffer = d_indices;

    uint32_t triangleInputFlags[1] = {0};

    // in this example we have one SBT entry, and no per-primitive
    // materials:
    triangleInput.triangleArray.flags = triangleInputFlags;
    triangleInput.triangleArray.numSbtRecords = 1;
    triangleInput.triangleArray.sbtIndexOffsetBuffer = 0;
    triangleInput.triangleArray.sbtIndexOffsetSizeInBytes = 0;
    triangleInput.triangleArray.sbtIndexOffsetStrideInBytes = 0;

    // ==================================================================
    // BLAS setup
    // ==================================================================

    OptixAccelBuildOptions accelOptions = {};
    accelOptions.buildFlags = OPTIX_BUILD_FLAG_NONE
                              | OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
    accelOptions.motionOptions.numKeys = 1;
    accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

    OptixAccelBufferSizes blasBufferSizes;
    AST_OPTIX_CHECK(optixAccelComputeMemoryUsage
                    (m_optixContext,
                     &accelOptions,
                     &triangleInput,
                     1,  // num_build_inputs
                     &blasBufferSizes
                    ));

    // ==================================================================
    // prepare compaction
    // ==================================================================

    DeviceBuffer compactedSizeBuffer;
    compactedSizeBuffer.alloc(sizeof(uint64_t));

    OptixAccelEmitDesc emitDesc;
    emitDesc.type = OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
    emitDesc.result = (hipDeviceptr_t)compactedSizeBuffer.devicePtr();

    // ==================================================================
    // execute build (main stage)
    // ==================================================================

    DeviceBuffer tempBuffer;
    tempBuffer.alloc(blasBufferSizes.tempSizeInBytes);

    DeviceBuffer outputBuffer;
    outputBuffer.alloc(blasBufferSizes.outputSizeInBytes);

    AST_OPTIX_CHECK(optixAccelBuild(m_optixContext,
        /* stream */0,
                                &accelOptions,
                                &triangleInput,
                                1,
                                (hipDeviceptr_t)tempBuffer.devicePtr(),
                                tempBuffer.m_sizeInBytes,

                                (hipDeviceptr_t)outputBuffer.devicePtr(),
                                outputBuffer.m_sizeInBytes,

                                &asHandle,

                                &emitDesc, 1
    ));
    AST_CUDA_SYNC_CHECK();

    // ==================================================================
    // perform compaction
    // ==================================================================
    uint64_t compactedSize;
    compactedSizeBuffer.download(&compactedSize, 1);

    m_asBuffer.alloc(compactedSize);
    AST_OPTIX_CHECK(optixAccelCompact(m_optixContext,
        /*stream:*/0,
                                  asHandle,
                                  (hipDeviceptr_t)m_asBuffer.devicePtr(),
                                  m_asBuffer.m_sizeInBytes,
                                  &asHandle));
    AST_CUDA_SYNC_CHECK();

    // ==================================================================
    // aaaaaand .... clean up
    // ==================================================================
    outputBuffer.free(); // << the UNcompacted, temporary output buffer
    tempBuffer.free();
    compactedSizeBuffer.free();

    return asHandle;
}
